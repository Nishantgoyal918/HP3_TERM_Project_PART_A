
#include <hip/hip_runtime.h>
// TILE_SIZE and N are variable/parameter here
#define TILE_SIZE 4

__device__ void store_full_row(float* read_data,float* write_data,int i,int N)
{
    int global_y;
    int global_x = i*blockDim.x + threadIdx.x;
        
    global_y = 0*blockDim.y + threadIdx.y;
    write_data[global_y*N + global_x] = read_data[threadIdx.x + TILE_SIZE*global_y];
        
    global_y = 1*blockDim.y + threadIdx.y;
    write_data[global_y*N + global_x] = read_data[threadIdx.x + TILE_SIZE*global_y];
        
    global_y = 2*blockDim.y + threadIdx.y;
    write_data[global_y*N + global_x] = read_data[threadIdx.x + TILE_SIZE*global_y];
        
    global_y = 3*blockDim.y + threadIdx.y;
    write_data[global_y*N + global_x] = read_data[threadIdx.x + TILE_SIZE*global_y];
    
    __syncthreads();
}
__device__ void load_full_row(float* read_data,float* write_data,int i,int N)
{
    int global_y;
    int global_x = i*blockDim.x + threadIdx.x;
        
    global_y = 0*blockDim.y + threadIdx.y;
    write_data[threadIdx.x + TILE_SIZE*global_y] = read_data[global_y*N + global_x];
        
    global_y = 1*blockDim.y + threadIdx.y;
    write_data[threadIdx.x + TILE_SIZE*global_y] = read_data[global_y*N + global_x];
        
    global_y = 2*blockDim.y + threadIdx.y;
    write_data[threadIdx.x + TILE_SIZE*global_y] = read_data[global_y*N + global_x];
        
    global_y = 3*blockDim.y + threadIdx.y;
    write_data[threadIdx.x + TILE_SIZE*global_y] = read_data[global_y*N + global_x];
    
    __syncthreads();
}
__device__ void potrf_tile(float* t_A)
{
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
    __shared__ float temp2;
        
    if(t_x==t_y && t_x==0)
    {
        t_A[0*TILE_SIZE + 0] = sqrtf(t_A[0*TILE_SIZE + 0]);
        temp2 = t_A[0*TILE_SIZE + 0];
    }
    __syncthreads();
    if(t_x<t_y && t_x == 0)
    {
        t_A[t_y*TILE_SIZE + 0]/= temp2;
    }
    __syncthreads();
    if(0<t_y && 0<t_x && t_x<=t_y)
    {
        t_A[t_y*TILE_SIZE + t_x]-= t_A[t_x*TILE_SIZE + 0]*t_A[t_y*TILE_SIZE + 0];
    }
    __syncthreads();
        
    if(t_x==t_y && t_x==1)
    {
        t_A[1*TILE_SIZE + 1] = sqrtf(t_A[1*TILE_SIZE + 1]);
        temp2 = t_A[1*TILE_SIZE + 1];
    }
    __syncthreads();
    if(t_x<t_y && t_x == 1)
    {
        t_A[t_y*TILE_SIZE + 1]/= temp2;
    }
    __syncthreads();
    if(1<t_y && 1<t_x && t_x<=t_y)
    {
        t_A[t_y*TILE_SIZE + t_x]-= t_A[t_x*TILE_SIZE + 1]*t_A[t_y*TILE_SIZE + 1];
    }
    __syncthreads();
        
    if(t_x==t_y && t_x==2)
    {
        t_A[2*TILE_SIZE + 2] = sqrtf(t_A[2*TILE_SIZE + 2]);
        temp2 = t_A[2*TILE_SIZE + 2];
    }
    __syncthreads();
    if(t_x<t_y && t_x == 2)
    {
        t_A[t_y*TILE_SIZE + 2]/= temp2;
    }
    __syncthreads();
    if(2<t_y && 2<t_x && t_x<=t_y)
    {
        t_A[t_y*TILE_SIZE + t_x]-= t_A[t_x*TILE_SIZE + 2]*t_A[t_y*TILE_SIZE + 2];
    }
    __syncthreads();
        
    if(t_x==t_y && t_x==3)
    {
        t_A[3*TILE_SIZE + 3] = sqrtf(t_A[3*TILE_SIZE + 3]);
        temp2 = t_A[3*TILE_SIZE + 3];
    }
    __syncthreads();
    if(t_x<t_y && t_x == 3)
    {
        t_A[t_y*TILE_SIZE + 3]/= temp2;
    }
    __syncthreads();
    if(3<t_y && 3<t_x && t_x<=t_y)
    {
        t_A[t_y*TILE_SIZE + t_x]-= t_A[t_x*TILE_SIZE + 3]*t_A[t_y*TILE_SIZE + 3];
    }
    __syncthreads();
    
}
__device__ void trsm_tile(float *row_data,int i,int j,int N)
{
    int global_y = j*blockDim.y + threadIdx.y;
    int global_x = i*blockDim.x + threadIdx.x;
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
        
    if(t_x==0)
    {
        row_data[global_y*TILE_SIZE + t_x]/= row_data[global_x*TILE_SIZE + t_x];
    }
    __syncthreads();
    if(t_x > 0)
    {
        row_data[global_y*TILE_SIZE + t_x]-= row_data[global_x*TILE_SIZE +  0]*row_data[global_y*TILE_SIZE + 0];
    }
    __syncthreads();
        
    if(t_x==1)
    {
        row_data[global_y*TILE_SIZE + t_x]/= row_data[global_x*TILE_SIZE + t_x];
    }
    __syncthreads();
    if(t_x > 1)
    {
        row_data[global_y*TILE_SIZE + t_x]-= row_data[global_x*TILE_SIZE +  1]*row_data[global_y*TILE_SIZE + 1];
    }
    __syncthreads();
        
    if(t_x==2)
    {
        row_data[global_y*TILE_SIZE + t_x]/= row_data[global_x*TILE_SIZE + t_x];
    }
    __syncthreads();
    if(t_x > 2)
    {
        row_data[global_y*TILE_SIZE + t_x]-= row_data[global_x*TILE_SIZE +  2]*row_data[global_y*TILE_SIZE + 2];
    }
    __syncthreads();
        
    if(t_x==3)
    {
        row_data[global_y*TILE_SIZE + t_x]/= row_data[global_x*TILE_SIZE + t_x];
    }
    __syncthreads();
    if(t_x > 3)
    {
        row_data[global_y*TILE_SIZE + t_x]-= row_data[global_x*TILE_SIZE +  3]*row_data[global_y*TILE_SIZE + 3];
    }
    __syncthreads();
    
}
__device__ void syrk_tile(float* row_data,float* edit_data,int i,int j,int N) 
{
    int global_y = j*blockDim.y + threadIdx.y;
    int global_x = i*blockDim.x + threadIdx.x;
    int t_y = threadIdx.y;
    int t_x = threadIdx.x;
    float valueToSubtract = 0.0;
        
    valueToSubtract+= row_data[0 + global_y*TILE_SIZE]*row_data[0 + global_x*TILE_SIZE];
        
    valueToSubtract+= row_data[1 + global_y*TILE_SIZE]*row_data[1 + global_x*TILE_SIZE];
        
    valueToSubtract+= row_data[2 + global_y*TILE_SIZE]*row_data[2 + global_x*TILE_SIZE];
        
    valueToSubtract+= row_data[3 + global_y*TILE_SIZE]*row_data[3 + global_x*TILE_SIZE];
        
    edit_data[t_y*TILE_SIZE + t_x]-= valueToSubtract;
    __syncthreads();
}
__device__ void store_zeros(float* A,int N)
{
    int t_y = threadIdx.y;
    int t_x = threadIdx.x;
                
    A[1*blockDim.x + t_x + (0*blockDim.y + t_y)*N] = 0.0;
            
    A[2*blockDim.x + t_x + (0*blockDim.y + t_y)*N] = 0.0;
            
    A[3*blockDim.x + t_x + (0*blockDim.y + t_y)*N] = 0.0;
                        
    A[2*blockDim.x + t_x + (1*blockDim.y + t_y)*N] = 0.0;
            
    A[3*blockDim.x + t_x + (1*blockDim.y + t_y)*N] = 0.0;
                        
    A[3*blockDim.x + t_x + (2*blockDim.y + t_y)*N] = 0.0;
            
    __syncthreads();
}
