#include "right_looking.cu"
int main()
{
    int n,N;
    printf("Enter dimension (N) : ");
    scanf("%d",&n);
    if((n%TILE_SIZE)==0)
        N = n;
    else
        N = (((int) (n/TILE_SIZE)) + 1)*TILE_SIZE;
    size_t size = N*N*sizeof(float);
    float *M = (float *)malloc(size);
    if(M == NULL)
    {
        fprintf(stderr,"Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    int i,j;
    printf("Enter input matrix: \n");
    for(i=0;i<N;i++)
    {
        for(j=0;j<N;j++)
        {
            if(i>=n || j>=n)
                M[i*N + j] = 1;     //Padding the matrix with 1
            else
                scanf("%f",&M[i*N + j]);
        }
    }
    hipError_t err = hipSuccess;
    float *read_data = NULL;
    err = hipMalloc((void **)&read_data,N*N*sizeof(float));
    if(err != hipSuccess)
    {
        fprintf(stderr,"Failed to allocate matrix on the CUDA device! (error code %s)\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Coping the matrix from host memory to device memory\n");
    err = hipMemcpy(read_data,M,size,hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr,"Failed to copy matrix from host to device (error code %s)\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Testing for matrix M [%dx%d]\n",N,N);
    dim3 grid(1,1,1);
    dim3 block(TILE_SIZE,TILE_SIZE,1);
    right_looking_launch_kernel<<<grid,block>>>(read_data,N);
    err = hipMemcpy(M,read_data,size,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy the output matrix M from device to Host (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Printing output matrix\n");
    for(i=0;i<n;i++)
    {
        for(j=0;j<n;j++)
        {
            if(j<=i)
                printf("%f\t",M[i*N + j]);
            else
                printf("%f\t",0.0);
        }
        printf("\n");
    }
    err = hipFree(read_data);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipDeviceReset();
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the CUDA device (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    free(M);
    printf("DONE!\n");
    return 0;
}