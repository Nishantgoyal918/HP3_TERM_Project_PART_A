#include "hip/hip_runtime.h"
#include "./headers.h"
#include "./left_looking_kernel.cu"

int main()
{

    FILE *fptr;
    fptr = fopen("./input.txt", "r");


    hipError_t err = hipSuccess;

    int devCount;
    hipGetDeviceCount(&devCount);

    hipDeviceProp_t devp;
    hipGetDeviceProperties(&devp, 0);

    int INPUT_SIZE = 0;
    fscanf(fptr, "%d", &INPUT_SIZE);

    size_t size = INPUT_SIZE * INPUT_SIZE * (sizeof(float));
    printf("Testing for matrix M [%dx%d]\n", INPUT_SIZE, INPUT_SIZE);

    float *M = (float *)malloc(size);

    if(M == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    printf("Reading input matrix: \n");
    for(int i=0; i<INPUT_SIZE; i++)
    {
        for(int j=0; j<INPUT_SIZE; j++)
        {
            fscanf(fptr, "%f ", &M[i * INPUT_SIZE + j]);
        }
    }

    // printf("Printing input matrix\n");
    // for(int i=0; i<INPUT_SIZE; i++)
    // {
    //     for(int j=0; j<INPUT_SIZE; j++)
    //     {
    //         printf("%f ", M[i * INPUT_SIZE + j]);
    //     }
    //     printf("\n");
    // }

    printf("\n\n");

    float *d_M = NULL;
    err = hipMalloc((void **)&d_M, size);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate matrix M on the CUDA device! (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    printf("Copy the matrix M from host memory to CUDA device\n\n");

    err = hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix M from host to device (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 grid(1, 1, 1);
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    // no of tiles in a column
    int no_of_tiles = (INPUT_SIZE / TILE_SIZE) + (INPUT_SIZE % TILE_SIZE != 0); // ceil of (INPUT_SIZE / TILE_SIZE)

    if(TILE_SIZE == INPUT_SIZE)
    {
        left_looking_kernel<<<grid, block, 1 * TILE_SIZE * (TILE_SIZE + 1) * sizeof(float)>>>(d_M, INPUT_SIZE);
    }
    else if((no_of_tiles + 2) * TILE_SIZE * (TILE_SIZE + 1) * sizeof(float) < devp.sharedMemPerBlock)
    {
        left_looking_kernel<<<grid, block, (no_of_tiles + 2) * TILE_SIZE * (TILE_SIZE + 1) * sizeof(float)>>>(d_M, INPUT_SIZE);
    }
    else
    {
        left_looking_kernel_less_mem<<<grid, block, 4 * TILE_SIZE * (TILE_SIZE + 1) * sizeof(float)>>>(d_M, INPUT_SIZE);
    }
    err = hipGetLastError();

    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CUDA kernel (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(M, d_M, size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy the output matrix M from device to Host (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Printing output matrix\n");
    // for(int i=0; i<INPUT_SIZE; i++)
    // {
    //     for(int j=0; j<INPUT_SIZE; j++)
    //     {
    //         printf("%f ", M[i * INPUT_SIZE + j]);
    //     }
    //     printf("\n");
    // }

    err = hipFree(d_M);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(M);

    err = hipDeviceReset();
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the CUDA device (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("DONE!\n");

}